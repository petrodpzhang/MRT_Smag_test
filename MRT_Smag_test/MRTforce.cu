#include "hip/hip_runtime.h"
#include "MRTforce.cuh"
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <cmath>
#include <fstream>
#include <sstream>
#include <string>

using namespace std;

const int Q = 19;
__constant__ int cx[19];
__constant__ int cy[19];
__constant__ int cz[19];
__constant__ double w[19];
__constant__ int N[3];

//1.initialize
void MRTGPU::init(LBMpara params)
{
	Nx = params.Nx;
	Ny = params.Ny;
	Nz = params.Nz;
	rho0 = params.rho0;
	ux0 = params.ux0;
	uy0 = params.uy0;
	uz0 = params.uz0;
	gravity = params.gravity;
	int Nlattice = Nx * Ny * Nz;
	int QNlattice = Nx * Ny * Nz * Q;

	int _cx[19] = { 0,1,-1,0,0,0,0,1,-1,1,-1,1,-1,1,-1,0,0,0,0 };
	int _cy[19] = { 0,0,0,1,-1,0,0,1,1,-1,-1,0,0,0,0,1,-1,1,-1 };
	int _cz[19] = { 0,0,0,0,0,1,-1,0,0,0,0,1,1,-1,-1,1,1,-1,-1 };//Yu's setting
	double _w[19] = { 1.0 / 3.0,1.0 / 18.0,1.0 / 18.0,1.0 / 18.0,1.0 / 18.0,1.0 / 18.0,1.0 / 18.0,1.0 / 36.0,1.0 / 36.0,1.0 / 36.0,
		1.0 / 36.0,1.0 / 36.0,1.0 / 36.0,1.0 / 36.0,1.0 / 36.0,1.0 / 36.0,1.0 / 36.0,1.0 / 36.0,1.0 / 36.0 };
	int _N[3] = { Nx,Ny,Nz };

	hipMemcpyToSymbol(HIP_SYMBOL(cx), _cx, sizeof(int) * Q);
	hipMemcpyToSymbol(HIP_SYMBOL(cy), _cy, sizeof(int) * Q);
	hipMemcpyToSymbol(HIP_SYMBOL(cz), _cz, sizeof(int) * Q);
	hipMemcpyToSymbol(HIP_SYMBOL(w), _w, sizeof(double) * Q);
	hipMemcpyToSymbol(HIP_SYMBOL(N), _N, sizeof(int) * 3);

	int threadsAlongX = 8;
	int threadsAlongY = 8;
	int threadsAlongZ = 8;

	block = dim3(threadsAlongX, threadsAlongY, threadsAlongZ);
	grid = dim3(1 + (Nx - 1) / threadsAlongX, 1 + (Ny - 1) / threadsAlongY, 1 + (Nz - 1) / threadsAlongZ);

	// allocate memory on CPU and GPU 
	h_geo = (int*)malloc(sizeof(int) * Nlattice);
	h_ux = (double*)malloc(sizeof(double) * Nlattice);
	h_uy = (double*)malloc(sizeof(double) * Nlattice);
	h_uz = (double*)malloc(sizeof(double) * Nlattice);
	h_rho = (double*)malloc(sizeof(double) * Nlattice);
	h_f = (double*)malloc(sizeof(double) * Nlattice * Q);
	h_f_post = (double*)malloc(sizeof(double) * Nlattice * Q);

	output_rho = (double*)malloc(sizeof(double) * Nlattice);
	output_ux = (double*)malloc(sizeof(double) * Nlattice);
	output_uy = (double*)malloc(sizeof(double) * Nlattice);
	output_uz = (double*)malloc(sizeof(double) * Nlattice);
	test = (double*)malloc(sizeof(double) * Nlattice);

	hipMalloc((void**)&d_geo, Nlattice * sizeof(int));
	hipMalloc((void**)&d_f, sizeof(double) * Nlattice * Q);
	hipMalloc((void**)&d_f_post, sizeof(double) * Nlattice * Q);
	hipMalloc((void**)&d_feq, sizeof(double) * Nlattice * Q);
	hipMalloc((void**)&m_f, sizeof(double) * Nlattice * Q);
	hipMalloc((void**)&m_eq, sizeof(double) * Nlattice * Q);
	hipMalloc((void**)&inv_f, sizeof(double) * Nlattice * Q);
	hipMalloc((void**)&inv_feq, sizeof(double) * Nlattice * Q);

	hipMalloc((void**)&d_rho, Nlattice * sizeof(double));
	hipMalloc((void**)&d_ux, Nlattice * sizeof(double));
	hipMalloc((void**)&d_uy, Nlattice * sizeof(double));
	hipMalloc((void**)&d_uz, Nlattice * sizeof(double));

	hipMalloc((void**)&s_xx, Nlattice * sizeof(double));
	hipMalloc((void**)&s_xy, Nlattice * sizeof(double));
	hipMalloc((void**)&s_xz, Nlattice * sizeof(double));
	hipMalloc((void**)&s_yx, Nlattice * sizeof(double));
	hipMalloc((void**)&s_yy, Nlattice * sizeof(double));
	hipMalloc((void**)&s_yz, Nlattice * sizeof(double));
	hipMalloc((void**)&s_zx, Nlattice * sizeof(double));
	hipMalloc((void**)&s_zy, Nlattice * sizeof(double));
	hipMalloc((void**)&s_zz, Nlattice * sizeof(double));
	hipMalloc((void**)&sigma, Nlattice * sizeof(double));

	ifstream inf("D:\\flowAroundObst.dat"); //read the geo file

	int i = 0;

	while (inf >> h_geo[i])
		++i;
	inf.close();

	for (int z = 0; z < Nz; z++)
	{
		for (int y = 0; y < Ny; y++)
		{
			for (int x = 0; x < Nx; x++)
			{
				int index = z * Nx * Ny + y * Nx + x;

				if (h_geo[index] == 0 || h_geo[index] == 4)
				{
					h_ux[index] = 0.0;
					h_uy[index] = 0.0;
					h_uz[index] = 0.0;// uz0 - gravity / 2;
					h_rho[index] = rho0;
				}
				else if (h_geo[index] == 3)
				{
					h_ux[index] = 0.0;
					h_uy[index] = 0.0;
					h_uz[index] = uz0;
					h_rho[index] = rho0;
				}
				else if (h_geo[index] == 1)
				{
					h_ux[index] = 0.0;
					h_uy[index] = 0.0;
					h_uz[index] = 0.0;
					h_rho[index] = 0.0;
				}
				else if (h_geo[index] == 2)
				{
					h_ux[index] = 0.0;
					h_uy[index] = 0.0;
					h_uz[index] = 0.0;// uz0 - gravity / 2;
					h_rho[index] = rho0;
				}
			}
		}
	}
	for (int z = 0; z < Nz; z++)
	{
		for (int y = 0; y < Ny; y++)
		{
			for (int x = 0; x < Nx; x++)
			{
				int index = z * Nx * Ny + y * Nx + x;

				for (int q = 0; q < Q; q++)
				{
					if (h_geo[index] == 0 || h_geo[index] == 2 || h_geo[index] == 3 || h_geo[index] == 4)
					{
						h_f[Nlattice * q + index] = _w[q] * h_rho[index] * (1.0 + 3.0 * (_cx[q] * h_ux[index] + _cy[q] * h_uy[index] + _cz[q] * h_uz[index])
							+ 4.5 * (_cx[q] * h_ux[index] + _cy[q] * h_uy[index] + _cz[q] * h_uz[index]) * (_cx[q] * h_ux[index] + _cy[q] * h_uy[index] + _cz[q] * h_uz[index])
							- 1.5 * (h_ux[index] * h_ux[index] + h_uy[index] * h_uy[index] + h_uz[index] * h_uz[index]));
						h_f_post[Nlattice * q + index] = 0.0;
					}
					else if (h_geo[index] == 1)
					{
						h_f[Nlattice * q + index] = 0.0;
						h_f_post[Nlattice * q + index] = 0.0;
					}
				}
			}
		}
	}
	hipMemcpy(d_geo, h_geo, Nlattice * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_f, h_f, QNlattice * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_f_post, h_f_post, QNlattice * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_ux, h_ux, Nlattice * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_uy, h_uy, Nlattice * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_uz, h_uz, Nlattice * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_rho, h_rho, Nlattice * sizeof(double), hipMemcpyHostToDevice);
}

////////////////////////////////////////////////2.gpu side functions//////////////////////////////////////

__global__ void kernelfeq(int* __restrict__ d_geo, double* __restrict__ d_feq,
	double* __restrict__ d_rho, double* __restrict__ d_ux, double* __restrict__ d_uy, double* __restrict__ d_uz);
void MRTGPU::feq()
{
	kernelfeq << <grid, block >> > (d_geo, d_feq, d_rho, d_ux, d_uy, d_uz);
	hipDeviceSynchronize();
}
__global__ void kernelfeq(int* __restrict__ d_geo, double* __restrict__ d_feq,
	double* __restrict__ d_rho, double* __restrict__ d_ux, double* __restrict__ d_uy, double* __restrict__ d_uz)
{
	const double rho0 = 1.0;
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int index = k * N[0] * N[1] + j * N[0] + i;
	int Nlattice = N[0] * N[1] * N[2];

	for (int q = 0; q < 19; q++)
	{
		if (d_geo[index] == 0 || d_geo[index] == 2 || d_geo[index] == 3 || d_geo[index] == 4)
			d_feq[index + Nlattice * q] = w[q] * (d_rho[index] + rho0 * (3.0 * (cx[q] * d_ux[index] + cy[q] * d_uy[index] + cz[q] * d_uz[index])
				+ 4.5 * (cx[q] * d_ux[index] + cy[q] * d_uy[index] + cz[q] * d_uz[index]) * (cx[q] * d_ux[index] + cy[q] * d_uy[index] + cz[q] * d_uz[index])
				- 1.5 * (d_ux[index] * d_ux[index] + d_uy[index] * d_uy[index] + d_uz[index] * d_uz[index])));
		else if (d_geo[index] == 1)
			d_feq[index + Nlattice * q] = 0.0;
	}
}

///////////////////////////////////////////////////////////////////////////////////

__global__ void kernelrate_strain(int* __restrict__ d_geo, double* __restrict__ d_feq, double* __restrict__ d_f,
	double* __restrict__ s_xx, double* __restrict__ s_xy, double* __restrict__ s_xz,
	double* __restrict__ s_yx, double* __restrict__ s_yy, double* __restrict__ s_yz,
	double* __restrict__ s_zx, double* __restrict__ s_zy, double* __restrict__ s_zz, double* __restrict__ sigma);
void MRTGPU::rate_strain()
{
	kernelrate_strain << <grid, block >> > (d_geo, d_feq, d_f, s_xx, s_xy, s_xz, s_yx, s_yy, s_yz, s_zx, s_zy, s_zz, sigma);
	hipDeviceSynchronize();
}
__global__ void kernelrate_strain(int* __restrict__ d_geo, double* __restrict__ d_feq, double* __restrict__ d_f,
	double* __restrict__ s_xx, double* __restrict__ s_xy, double* __restrict__ s_xz,
	double* __restrict__ s_yx, double* __restrict__ s_yy, double* __restrict__ s_yz,
	double* __restrict__ s_zx, double* __restrict__ s_zy, double* __restrict__ s_zz, double* __restrict__ sigma)
{
	const int Q = 19;
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int index = k * N[0] * N[1] + j * N[0] + i;
	int Nlattice = N[0] * N[1] * N[2];

	s_xx[index] = 0.0; s_xy[index] = 0.0; s_xz[index] = 0.0;
	s_yx[index] = 0.0; s_yy[index] = 0.0; s_yz[index] = 0.0;
	s_zx[index] = 0.0; s_zy[index] = 0.0; s_zz[index] = 0.0;

	for (int q = 0; q < Q; q++)
	{
		s_xx[index] += (d_f[index + Nlattice * q] - d_feq[index + Nlattice * q]) * cx[q] * cx[q];
		s_xy[index] += (d_f[index + Nlattice * q] - d_feq[index + Nlattice * q]) * cx[q] * cy[q];
		s_xz[index] += (d_f[index + Nlattice * q] - d_feq[index + Nlattice * q]) * cx[q] * cz[q];
		s_yx[index] += (d_f[index + Nlattice * q] - d_feq[index + Nlattice * q]) * cx[q] * cy[q];
		s_yy[index] += (d_f[index + Nlattice * q] - d_feq[index + Nlattice * q]) * cy[q] * cy[q];
		s_yz[index] += (d_f[index + Nlattice * q] - d_feq[index + Nlattice * q]) * cy[q] * cz[q];
		s_zx[index] += (d_f[index + Nlattice * q] - d_feq[index + Nlattice * q]) * cx[q] * cz[q];
		s_zy[index] += (d_f[index + Nlattice * q] - d_feq[index + Nlattice * q]) * cy[q] * cz[q];
		s_zz[index] += (d_f[index + Nlattice * q] - d_feq[index + Nlattice * q]) * cz[q] * cz[q];
	}
	if (d_geo[index] == 0 || d_geo[index] == 2 || d_geo[index] == 3 || d_geo[index] == 4)
		sigma[index] = sqrt(2 * (s_xx[index] * s_xx[index] + s_xy[index] * s_xy[index] + s_xz[index] * s_xz[index]
			+ s_yx[index] * s_yx[index] + s_yy[index] * s_yy[index] + s_yz[index] * s_yz[index]
			+ s_zx[index] * s_zx[index] + s_zy[index] * s_zy[index] + s_zz[index] * s_zz[index]));
	else if (d_geo[index] == 1)
		sigma[index] = 0.0;
}

///////////////////////////////////////////////////////////////////////////////////

__global__ void kernelmf_meq(int* __restrict__ d_geo, double* __restrict__ d_f, double* __restrict__ m_f, double* __restrict__ m_eq,
	double* __restrict__ sigma, double* __restrict__ d_rho, double* __restrict__ d_ux, double* __restrict__ d_uy, double* __restrict__ d_uz);
void MRTGPU::mf_meq()
{
	kernelmf_meq << <grid, block >> > (d_geo, d_f, m_f, m_eq, sigma, d_rho, d_ux, d_uy, d_uz);
	hipDeviceSynchronize();
}
__global__ void kernelmf_meq(int* __restrict__ d_geo, double* __restrict__ d_f, double* __restrict__ m_f, double* __restrict__ m_eq,
	double* __restrict__ sigma, double* __restrict__ d_rho, double* __restrict__ d_ux, double* __restrict__ d_uy, double* __restrict__ d_uz)
{
	const double rho0 = 1.0;
	const double tau = 0.55;/////////////calculaton
	const double C_Smag = 0.16;
	const int Q = 19;

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int index = k * N[0] * N[1] + j * N[0] + i;
	int Nlattice = N[0] * N[1] * N[2];

	const double S_diag[19] = { 0.0, 1.19,1.4,0.0,1.2,0.0,1.2,0.0,1.2,1.0,1.4,1.0,1.4,1.0,1.0,1.0,1.98,1.98,1.98 };
	const double F_diag[19] = { 19.0,2394.0,252.0,10.0,40.0,10.0,40.0,10.0,40.0,36.0,72.0,12.0,24.0,4.0,4.0,4.0,8.0,8.0,8.0 };

	const double mm[19][19] = {
	{1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0},
	{-30.0,-11.0,-11.0,-11.0,-11.0,-11.0,-11.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0},
	{12.0,-4.0,-4.0,-4.0,-4.0,-4.0,-4.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0},
	{0.0,1.0,-1.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0,1.0,-1.0,1.0,-1.0,0.0,0.0,0.0,0.0},
	{0.0,-4.0,4.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0,1.0,-1.0,1.0,-1.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,1.0,-1.0,0.0,0.0,1.0,1.0,-1.0,-1.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0},
	{0.0,0.0,0.0,-4.0,4.0,0.0,0.0,1.0,1.0,-1.0,-1.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0},
	{0.0,0.0,0.0,0.0,0.0,1.0,-1.0,0.0,0.0,0.0,0.0,1.0,1.0,-1.0,-1.0,1.0,1.0,-1.0,-1.0},
	{0.0,0.0,0.0,0.0,0.0,-4.0,4.0,0.0,0.0,0.0,0.0,1.0,1.0,-1.0,-1.0,1.0,1.0,-1.0,-1.0},
	{0.0,2.0,2.0,-1.0,-1.0,-1.0,-1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,-2.0,-2.0,-2.0,-2.0},
	{0.0,-4.0,-4.0,2.0,2.0,2.0,2.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,-2.0,-2.0,-2.0,-2.0},
	{0.0,0.0,0.0,1.0,1.0,-1.0,-1.0,1.0,1.0,1.0,1.0,-1.0,-1.0,-1.0,-1.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,-2.0,-2.0,2.0,2.0,1.0,1.0,1.0,1.0,-1.0,-1.0,-1.0,-1.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,1.0,-1.0,-1.0,1.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,1.0,-1.0,-1.0,1.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,1.0,-1.0,-1.0,1.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0,-1.0,1.0,-1.0,1.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,-1.0,-1.0,1.0,1.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,1.0,1.0,-1.0,-1.0,-1.0,-1.0,1.0,1.0} };

	for (int q = 0; q < Q; q++)
	{
		m_f[index + Nlattice * q] = 0.0;
	}
	if (d_geo[index] == 0 || d_geo[index] == 2 || d_geo[index] == 3 || d_geo[index] == 4)//geo==1
	{
		for (int q = 0; q < Q; q++)
		{
			for (int qq = 0; qq < Q; qq++)
			{
				if (q == 0 || q == 1 || q == 2 || q == 3 || q == 4 || q == 5 || q == 6 || q == 7 || q == 8 || q == 10 || q == 12 || q == 16 || q == 17 || q == 18)
					m_f[index + Nlattice * q] += S_diag[q] / F_diag[q] * mm[q][qq] * d_f[index + Nlattice * qq];
				else if (q == 9 || q == 11 || q == 13 || q == 14 || q == 15)
					m_f[index + Nlattice * q] += mm[q][qq] * d_f[index + Nlattice * qq] / (0.5 * (sqrt(tau * tau + 18.0 * C_Smag * C_Smag * sigma[index]) + tau)) / F_diag[q];
			}//tau_t = 0.5*(pow(pow(tau,2) + 18.0*pow(C_Smagorinsky,2)*sigma[index],0.5) - tau);
		}
		m_eq[index + Nlattice * 0] = S_diag[0] / F_diag[0] * d_rho[index];
		m_eq[index + Nlattice * 1] = S_diag[1] / F_diag[1] * ((-11.0) * d_rho[index] + 19.0 * (d_ux[index] * d_ux[index] + d_uy[index] * d_uy[index] + d_uz[index] * d_uz[index]));
		m_eq[index + Nlattice * 2] = S_diag[2] / F_diag[2] * (-475.0 / 63.0) * (d_ux[index] * d_ux[index] + d_uy[index] * d_uy[index] + d_uz[index] * d_uz[index]);
		m_eq[index + Nlattice * 3] = S_diag[3] / F_diag[3] * d_ux[index];
		m_eq[index + Nlattice * 4] = S_diag[4] / F_diag[4] * (-2.0 / 3.0) * d_ux[index];
		m_eq[index + Nlattice * 5] = S_diag[5] / F_diag[5] * d_uy[index];
		m_eq[index + Nlattice * 6] = S_diag[6] / F_diag[6] * (-2.0 / 3.0) * d_uy[index];
		m_eq[index + Nlattice * 7] = S_diag[7] / F_diag[7] * d_uz[index];
		m_eq[index + Nlattice * 8] = S_diag[8] / F_diag[8] * (-2.0 / 3.0) * d_uz[index];
		m_eq[index + Nlattice * 9] = 2.0 * d_ux[index] * d_ux[index] - (d_uy[index] * d_uy[index] + d_uz[index] * d_uz[index])
			/ (0.5 * (sqrt(tau * tau + 18.0 * C_Smag * C_Smag * sigma[index]) + tau)) / F_diag[9];
		m_eq[index + Nlattice * 10] = 0.0;
		m_eq[index + Nlattice * 11] = (d_uy[index] * d_uy[index] - d_uz[index] * d_uz[index])
			/ (0.5 * (sqrt(tau * tau + 18.0 * C_Smag * C_Smag * sigma[index]) + tau)) / F_diag[11];
		m_eq[index + Nlattice * 12] = 0.0;
		m_eq[index + Nlattice * 13] = d_ux[index] * d_uy[index]
			/ (0.5 * (sqrt(tau * tau + 18.0 * C_Smag * C_Smag * sigma[index]) + tau)) / F_diag[13];
		m_eq[index + Nlattice * 14] = d_uy[index] * d_uz[index]
			/ (0.5 * (sqrt(tau * tau + 18.0 * C_Smag * C_Smag * sigma[index]) + tau)) / F_diag[14];
		m_eq[index + Nlattice * 15] = d_ux[index] * d_uz[index]
			/ (0.5 * (sqrt(tau * tau + 18.0 * C_Smag * C_Smag * sigma[index]) + tau)) / F_diag[15];
		m_eq[index + Nlattice * 16] = 0.0;
		m_eq[index + Nlattice * 17] = 0.0;
		m_eq[index + Nlattice * 18] = 0.0;
	}
	if (d_geo[index] == 1)
	{
		for (int q = 0; q < Q; q++)
		{
			m_eq[index + Nlattice * q] = 0.0;
		}
	}
}

//////////////////////////////////////////////////////////////////////////////////

__global__ void kernelInvf_feq(int* __restrict__ d_geo, double* __restrict__ m_f, double* __restrict__ m_eq,
	double* __restrict__ inv_f, double* __restrict__ inv_feq);
void MRTGPU::Invf_feq()
{
	kernelInvf_feq << <grid, block >> > (d_geo, m_f, m_eq, inv_f, inv_feq);
	hipDeviceSynchronize();
}
__global__ void kernelInvf_feq(int* __restrict__ d_geo, double* __restrict__ m_f, double* __restrict__ m_eq,
	double* __restrict__ inv_f, double* __restrict__ inv_feq)
{
	const int Q = 19;
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int index = k * N[0] * N[1] + j * N[0] + i;
	int Nlattice = N[0] * N[1] * N[2];

	const double mmt[19][19] = {
	{1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0},
	{-30.0,-11.0,-11.0,-11.0,-11.0,-11.0,-11.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0,8.0},
	{12.0,-4.0,-4.0,-4.0,-4.0,-4.0,-4.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0},
	{0.0,1.0,-1.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0,1.0,-1.0,1.0,-1.0,0.0,0.0,0.0,0.0},
	{0.0,-4.0,4.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0,1.0,-1.0,1.0,-1.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,1.0,-1.0,0.0,0.0,1.0,1.0,-1.0,-1.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0},
	{0.0,0.0,0.0,-4.0,4.0,0.0,0.0,1.0,1.0,-1.0,-1.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0},
	{0.0,0.0,0.0,0.0,0.0,1.0,-1.0,0.0,0.0,0.0,0.0,1.0,1.0,-1.0,-1.0,1.0,1.0,-1.0,-1.0},
	{0.0,0.0,0.0,0.0,0.0,-4.0,4.0,0.0,0.0,0.0,0.0,1.0,1.0,-1.0,-1.0,1.0,1.0,-1.0,-1.0},
	{0.0,2.0,2.0,-1.0,-1.0,-1.0,-1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,-2.0,-2.0,-2.0,-2.0},
	{0.0,-4.0,-4.0,2.0,2.0,2.0,2.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,-2.0,-2.0,-2.0,-2.0},
	{0.0,0.0,0.0,1.0,1.0,-1.0,-1.0,1.0,1.0,1.0,1.0,-1.0,-1.0,-1.0,-1.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,-2.0,-2.0,2.0,2.0,1.0,1.0,1.0,1.0,-1.0,-1.0,-1.0,-1.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,1.0,-1.0,-1.0,1.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,1.0,-1.0,-1.0,1.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,1.0,-1.0,-1.0,1.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0,-1.0,1.0,-1.0,1.0,0.0,0.0,0.0,0.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,-1.0,-1.0,1.0,1.0,0.0,0.0,0.0,0.0,1.0,-1.0,1.0,-1.0},
	{0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,1.0,1.0,-1.0,-1.0,-1.0,-1.0,1.0,1.0} };

	for (int q = 0; q < Q; q++)
	{
		inv_f[index + Nlattice * q] = 0.0;
		inv_feq[index + Nlattice * q] = 0.0;
	}
	if (d_geo[index] == 0 || d_geo[index] == 2 || d_geo[index] == 3 || d_geo[index] == 4)//geo==1
	{
		for (int q = 0; q < Q; q++)
		{
			for (int qq = 0; qq < Q; qq++)
			{
				inv_f[index + Nlattice * q] += mmt[qq][q] * m_f[index + Nlattice * qq];
				inv_feq[index + Nlattice * q] += mmt[qq][q] * m_eq[index + Nlattice * qq];
			}
		}
	}
}

//////////////////////////////////////////////////////////////////////////////////

__global__ void kernelcollision(int* __restrict__ d_geo, double* __restrict__ inv_f, double* __restrict__ inv_feq,
	double* __restrict__ d_f, double* __restrict__ d_f_post);
void MRTGPU::collision()
{
	kernelcollision << <grid, block >> > (d_geo, inv_f, inv_feq, d_f, d_f_post);
	hipDeviceSynchronize();
}
__global__ void kernelcollision(int* __restrict__ d_geo, double* __restrict__ inv_f, double* __restrict__ inv_feq,
	double* __restrict__ d_f, double* __restrict__ d_f_post)
{
	const int Q = 19;
	int indexf[19];
	const double gravity = 0.00098;

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int index = k * N[0] * N[1] + j * N[0] + i;
	int Nlattice = N[0] * N[1] * N[2];

	if (d_geo[index] == 0 || d_geo[index] == 1 || d_geo[index] == 2 || d_geo[index] == 3 || d_geo[index] == 4)
	{
		int i_0 = (i + 0 + N[0]) % N[0];
		int j_0 = (j + 0 + N[1]) % N[1];
		int k_0 = (k + 0 + N[2]) % N[2];
		int index0 = k_0 * N[0] * N[1] + j_0 * N[0] + i_0;
		d_f_post[index0 + Nlattice * 0] = d_f[index + Nlattice * 0] - (inv_f[index + Nlattice * 0] - inv_feq[index + Nlattice * 0]);

		int i_1 = (i + 1 + N[0]) % N[0];
		int j_1 = (j + 0 + N[1]) % N[1];
		int k_1 = (k + 0 + N[2]) % N[2];
		int index1 = k_1 * N[0] * N[1] + j_1 * N[0] + i_1;
		d_f_post[index1 + Nlattice * 1] = d_f[index + Nlattice * 1] - (inv_f[index + Nlattice * 1] - inv_feq[index + Nlattice * 1]);

		int i_2 = (i - 1 + N[0]) % N[0];
		int j_2 = (j + 0 + N[1]) % N[1];
		int k_2 = (k + 0 + N[2]) % N[2];
		int index2 = k_2 * N[0] * N[1] + j_2 * N[0] + i_2;
		d_f_post[index2 + Nlattice * 2] = d_f[index + Nlattice * 2] - (inv_f[index + Nlattice * 2] - inv_feq[index + Nlattice * 2]);

		int i_3 = (i + 0 + N[0]) % N[0];
		int j_3 = (j + 1 + N[1]) % N[1];
		int k_3 = (k + 0 + N[2]) % N[2];
		int index3 = k_3 * N[0] * N[1] + j_3 * N[0] + i_3;
		d_f_post[index3 + Nlattice * 3] = d_f[index + Nlattice * 3] - (inv_f[index + Nlattice * 3] - inv_feq[index + Nlattice * 3]);

		int i_4 = (i + 0 + N[0]) % N[0];
		int j_4 = (j - 1 + N[1]) % N[1];
		int k_4 = (k + 0 + N[2]) % N[2];
		int index4 = k_4 * N[0] * N[1] + j_4 * N[0] + i_4;
		d_f_post[index4 + Nlattice * 4] = d_f[index + Nlattice * 4] - (inv_f[index + Nlattice * 4] - inv_feq[index + Nlattice * 4]);

		int i_5 = (i + 0 + N[0]) % N[0];
		int j_5 = (j + 0 + N[1]) % N[1];
		int k_5 = (k + 1 + N[2]) % N[2];
		int index5 = k_5 * N[0] * N[1] + j_5 * N[0] + i_5;
		d_f_post[index5 + Nlattice * 5] = d_f[index + Nlattice * 5] - (inv_f[index + Nlattice * 5] - inv_feq[index + Nlattice * 5]);

		int i_6 = (i + 0 + N[0]) % N[0];
		int j_6 = (j + 0 + N[1]) % N[1];
		int k_6 = (k - 1 + N[2]) % N[2];
		int index6 = k_6 * N[0] * N[1] + j_6 * N[0] + i_6;
		d_f_post[index6 + Nlattice * 6] = d_f[index + Nlattice * 6] - (inv_f[index + Nlattice * 6] - inv_feq[index + Nlattice * 6]);

		int i_7 = (i + 1 + N[0]) % N[0];
		int j_7 = (j + 1 + N[1]) % N[1];
		int k_7 = (k + 0 + N[2]) % N[2];
		int index7 = k_7 * N[0] * N[1] + j_7 * N[0] + i_7;
		d_f_post[index7 + Nlattice * 7] = d_f[index + Nlattice * 7] - (inv_f[index + Nlattice * 7] - inv_feq[index + Nlattice * 7]);

		int i_8 = (i - 1 + N[0]) % N[0];
		int j_8 = (j + 1 + N[1]) % N[1];
		int k_8 = (k + 0 + N[2]) % N[2];
		int index8 = k_8 * N[0] * N[1] + j_8 * N[0] + i_8;
		d_f_post[index8 + Nlattice * 8] = d_f[index + Nlattice * 8] - (inv_f[index + Nlattice * 8] - inv_feq[index + Nlattice * 8]);

		int i_9 = (i + 1 + N[0]) % N[0];
		int j_9 = (j - 1 + N[1]) % N[1];
		int k_9 = (k + 0 + N[2]) % N[2];
		int index9 = k_9 * N[0] * N[1] + j_9 * N[0] + i_9;
		d_f_post[index9 + Nlattice * 9] = d_f[index + Nlattice * 9] - (inv_f[index + Nlattice * 9] - inv_feq[index + Nlattice * 9]);

		int i_10 = (i - 1 + N[0]) % N[0];
		int j_10 = (j - 1 + N[1]) % N[1];
		int k_10 = (k + 0 + N[2]) % N[2];
		int index10 = k_10 * N[0] * N[1] + j_10 * N[0] + i_10;
		d_f_post[index10 + Nlattice * 10] = d_f[index + Nlattice * 10] - (inv_f[index + Nlattice * 10] - inv_feq[index + Nlattice * 10]);

		int i_11 = (i + 1 + N[0]) % N[0];
		int j_11 = (j + 0 + N[1]) % N[1];
		int k_11 = (k + 1 + N[2]) % N[2];
		int index11 = k_11 * N[0] * N[1] + j_11 * N[0] + i_11;
		d_f_post[index11 + Nlattice * 11] = d_f[index + Nlattice * 11] - (inv_f[index + Nlattice * 11] - inv_feq[index + Nlattice * 11]);

		int i_12 = (i - 1 + N[0]) % N[0];
		int j_12 = (j + 0 + N[1]) % N[1];
		int k_12 = (k + 1 + N[2]) % N[2];
		int index12 = k_12 * N[0] * N[1] + j_12 * N[0] + i_12;
		d_f_post[index12 + Nlattice * 12] = d_f[index + Nlattice * 12] - (inv_f[index + Nlattice * 12] - inv_feq[index + Nlattice * 12]);

		int i_13 = (i + 1 + N[0]) % N[0];
		int j_13 = (j + 0 + N[1]) % N[1];
		int k_13 = (k - 1 + N[2]) % N[2];
		int index13 = k_13 * N[0] * N[1] + j_13 * N[0] + i_13;
		d_f_post[index13 + Nlattice * 13] = d_f[index + Nlattice * 13] - (inv_f[index + Nlattice * 13] - inv_feq[index + Nlattice * 13]);

		int i_14 = (i - 1 + N[0]) % N[0];
		int j_14 = (j + 0 + N[1]) % N[1];
		int k_14 = (k - 1 + N[2]) % N[2];
		int index14 = k_14 * N[0] * N[1] + j_14 * N[0] + i_14;
		d_f_post[index14 + Nlattice * 14] = d_f[index + Nlattice * 14] - (inv_f[index + Nlattice * 14] - inv_feq[index + Nlattice * 14]);

		int i_15 = (i + 0 + N[0]) % N[0];
		int j_15 = (j + 1 + N[1]) % N[1];
		int k_15 = (k + 1 + N[2]) % N[2];
		int index15 = k_15 * N[0] * N[1] + j_15 * N[0] + i_15;
		d_f_post[index15 + Nlattice * 15] = d_f[index + Nlattice * 15] - (inv_f[index + Nlattice * 15] - inv_feq[index + Nlattice * 15]);

		int i_16 = (i + 0 + N[0]) % N[0];
		int j_16 = (j - 1 + N[1]) % N[1];
		int k_16 = (k + 1 + N[2]) % N[2];
		int index16 = k_16 * N[0] * N[1] + j_16 * N[0] + i_16;
		d_f_post[index16 + Nlattice * 16] = d_f[index + Nlattice * 16] - (inv_f[index + Nlattice * 16] - inv_feq[index + Nlattice * 16]);

		int i_17 = (i + 0 + N[0]) % N[0];
		int j_17 = (j + 1 + N[1]) % N[1];
		int k_17 = (k - 1 + N[2]) % N[2];
		int index17 = k_17 * N[0] * N[1] + j_17 * N[0] + i_17;
		d_f_post[index17 + Nlattice * 17] = d_f[index + Nlattice * 17] - (inv_f[index + Nlattice * 17] - inv_feq[index + Nlattice * 17]);

		int i_18 = (i + 0 + N[0]) % N[0];
		int j_18 = (j - 1 + N[1]) % N[1];
		int k_18 = (k - 1 + N[2]) % N[2];
		int index18 = k_18 * N[0] * N[1] + j_18 * N[0] + i_18;
		d_f_post[index18 + Nlattice * 18] = d_f[index + Nlattice * 18] - (inv_f[index + Nlattice * 18] - inv_feq[index + Nlattice * 18]);
	}
}

//////////////////////////////////////////////////////////////////////////////////

__global__ void kernelswap(int* __restrict__ d_geo, double* __restrict__ d_f, double* __restrict__ d_f_post);
void MRTGPU::swap()
{
	kernelswap << <grid, block >> > (d_geo, d_f, d_f_post);
	hipDeviceSynchronize();
}
__global__ void kernelswap(int* __restrict__ d_geo, double* __restrict__ d_f, double* __restrict__ d_f_post)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int index = k * N[0] * N[1] + j * N[0] + i;
	int Nlattice = N[0] * N[1] * N[2];

	for (int q = 0; q < 19; q++)
	{
		if (d_geo[index] == 0 || d_geo[index] == 1 || d_geo[index] == 2 || d_geo[index] == 3 || d_geo[index] == 4)
			d_f[index + Nlattice * q] = d_f_post[index + Nlattice * q];
	}
}

/////////////////////////////////////////////////////////////////////////////////

__global__ void kernelboundary(int* __restrict__ d_geo, double* __restrict__ d_f, double* __restrict__ d_rho);
void MRTGPU::boundary()
{
	kernelboundary << <grid, block >> > (d_geo, d_f, d_rho);
	hipDeviceSynchronize();
}
__global__ void kernelboundary(int* __restrict__ d_geo, double* __restrict__ d_f, double* __restrict__ d_rho)
{
	const double u0 = 0.02;// 0.02 - 0.00098 / 2;////////////////////////////
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int index = k * N[0] * N[1] + j * N[0] + i;
	int index_out = (k - 1) * N[0] * N[1] + j * N[0] + i;
	int Nlattice = N[0] * N[1] * N[2];
	int indexf[19];

	for (int q = 0; q < 19; q++)
	{
		int i_1 = (i + cx[q] + N[0]) % N[0];
		int j_1 = (j + cy[q] + N[1]) % N[1];
		int k_1 = (k + cz[q] + N[2]) % N[2];
		indexf[q] = k_1 * N[0] * N[1] + j_1 * N[0] + i_1;

		if (d_geo[index] == 2)
		{
			if (d_geo[indexf[q]] == 1)
			{
				if (q == 1) d_f[index + Nlattice * 2] = d_f[index + Nlattice * q];
				else if (q == 2) d_f[index + Nlattice * 1] = d_f[index + Nlattice * q];
				else if (q == 3) d_f[index + Nlattice * 4] = d_f[index + Nlattice * q];
				else if (q == 4) d_f[index + Nlattice * 3] = d_f[index + Nlattice * q];
				else if (q == 5) d_f[index + Nlattice * 6] = d_f[index + Nlattice * q];
				else if (q == 6) d_f[index + Nlattice * 5] = d_f[index + Nlattice * q];
				else if (q == 7) d_f[index + Nlattice * 10] = d_f[index + Nlattice * q];
				else if (q == 8) d_f[index + Nlattice * 9] = d_f[index + Nlattice * q];
				else if (q == 9) d_f[index + Nlattice * 8] = d_f[index + Nlattice * q];
				else if (q == 10) d_f[index + Nlattice * 7] = d_f[index + Nlattice * q];
				else if (q == 11) d_f[index + Nlattice * 14] = d_f[index + Nlattice * q];
				else if (q == 12) d_f[index + Nlattice * 13] = d_f[index + Nlattice * q];
				else if (q == 13) d_f[index + Nlattice * 12] = d_f[index + Nlattice * q];
				else if (q == 14) d_f[index + Nlattice * 11] = d_f[index + Nlattice * q];
				else if (q == 15) d_f[index + Nlattice * 18] = d_f[index + Nlattice * q];
				else if (q == 16) d_f[index + Nlattice * 17] = d_f[index + Nlattice * q];
				else if (q == 17) d_f[index + Nlattice * 16] = d_f[index + Nlattice * q];
				else if (q == 18) d_f[index + Nlattice * 15] = d_f[index + Nlattice * q];
			}
		}
		if (d_geo[index] == 3)
		{
			if (q == 6)
				d_f[index + Nlattice * 5] = d_f[index + Nlattice * q] + u0 * d_rho[index] / 3.0;
			else if (q == 13)
				d_f[index + Nlattice * 12] = d_f[index + Nlattice * q] + u0 * d_rho[index] / 6.0
				- 0.5 * (cx[12] * (d_f[index + Nlattice * 1] - d_f[index + Nlattice * 2])
					+ cy[12] * (d_f[index + Nlattice * 3] - d_f[index + Nlattice * 4]));
			else if (q == 14)
				d_f[index + Nlattice * 11] = d_f[index + Nlattice * q] + u0 * d_rho[index] / 6.0
				- 0.5 * (cx[11] * (d_f[index + Nlattice * 1] - d_f[index + Nlattice * 2])
					+ cy[11] * (d_f[index + Nlattice * 3] - d_f[index + Nlattice * 4]));
			else if (q == 17)
				d_f[index + Nlattice * 16] = d_f[index + Nlattice * q] + u0 * d_rho[index] / 6.0
				- 0.5 * (cx[16] * (d_f[index + Nlattice * 1] - d_f[index + Nlattice * 2])
					+ cy[16] * (d_f[index + Nlattice * 3] - d_f[index + Nlattice * 4]));
			else if (q == 18)
				d_f[index + Nlattice * 15] = d_f[index + Nlattice * q] + u0 * d_rho[index] / 6.0
				- 0.5 * (cx[15] * (d_f[index + Nlattice * 1] - d_f[index + Nlattice * 2])
					+ cy[15] * (d_f[index + Nlattice * 3] - d_f[index + Nlattice * 4]));
		}		
		if (d_geo[index] == 4)
		{
			if (q == 5)
				d_f[index + Nlattice * 6] = d_f[index_out + Nlattice * 6];
			else if (q == 11)
				d_f[index + Nlattice * 14] = d_f[index_out + Nlattice * 14];
			else if (q == 12)
				d_f[index + Nlattice * 13] = d_f[index_out + Nlattice * 13];
			else if (q == 15)
				d_f[index + Nlattice * 18] = d_f[index_out + Nlattice * 18];
			else if (q == 16)
				d_f[index + Nlattice * 17] = d_f[index_out + Nlattice * 17];
		}
	}
}

////////////////////////////////////////////////////////////////////////////////

__global__ void kernelmacroscopic(int* __restrict__ d_geo, double* __restrict__ d_f, double* __restrict__ inv_f, double* __restrict__ inv_feq,
	double* __restrict__ d_rho, double* __restrict__ d_ux, double* __restrict__ d_uy, double* __restrict__ d_uz);
void MRTGPU::macroscopic()
{
	kernelmacroscopic << <grid, block >> > (d_geo, d_f, inv_f, inv_feq, d_rho, d_ux, d_uy, d_uz);
	hipDeviceSynchronize();
}
__global__ void kernelmacroscopic(int* __restrict__ d_geo, double* __restrict__ d_f, double* __restrict__ inv_f, double* __restrict__ inv_feq,
	double* __restrict__ d_rho, double* __restrict__ d_ux, double* __restrict__ d_uy, double* __restrict__ d_uz)
{
	const int Q = 19;
	const double rho0 = 1.0;
	const double u0 = 0.02;
	const double gravity = 0.00098;
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int index = k * N[0] * N[1] + j * N[0] + i;
	int Nlattice = N[0] * N[1] * N[2];

	d_rho[index] = 0.0;
	d_ux[index] = 0.0; d_uy[index] = 0.0; d_uz[index] = 0.0;

	if (d_geo[index] == 0 || d_geo[index] == 4)
	{

		for (int q = 0; q < Q; q++)
		{
			d_rho[index] += d_f[index + Nlattice * q];
			d_ux[index] += cx[q] * d_f[index + Nlattice * q];
			d_uy[index] += cy[q] * d_f[index + Nlattice * q];
			d_uz[index] += cz[q] * d_f[index + Nlattice * q];
		}
		//d_uz[index] = d_uz[index] - 0.00049;
	}
	if (d_geo[index] == 3)
	{

		for (int q = 0; q < Q; q++)
			d_rho[index] += d_f[index + Nlattice * q];
		d_ux[index] = 0.0;
		d_uy[index] = 0.0;
		d_uz[index] = u0;
	}
	if (d_geo[index] == 2)
	{
		for (int q = 0; q < Q; q++)
			d_rho[index] += d_f[index + Nlattice * q];
		d_ux[index] = 0.0;
		d_uy[index] = 0.0;
		d_uz[index] = 0.0;
	}
	if (d_geo[index] == 1)
	{
		d_rho[index] = 0.0;
		d_ux[index] = 0.0;
		d_uy[index] = 0.0;
		d_uz[index] = 0.0;
	}
}

///////////////////////////////////////////////////////////////////////////////

void MRTGPU::output(int t)
{
	ofstream outputfile;
	stringstream sfile;
	sfile << "D:\\flowAroundObst-u002-t055-" << t << ".dat";
	string datafilename = sfile.str();

	outputfile.open(datafilename.c_str());
	outputfile << "TITLE = \"Case Data\"" << endl;
	outputfile << "VARIABLES = \"X\", \"Y\", \"Z\", \"geo\", \"Density\", \"Ux\", \"Uy\", \"Uz\"" << endl;
	outputfile << "ZONE I = 104, J = 104, K = 200, DATAPACKING = POINT" << endl;

	int Nlattice = Nx * Ny * Nz;
	hipMemcpy(output_rho, d_rho, Nlattice * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(output_ux, d_ux, Nlattice * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(output_uy, d_uy, Nlattice * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(output_uz, d_uz, Nlattice * sizeof(double), hipMemcpyDeviceToHost);

	for (int z = 0; z < Nz; z++)
	{
		for (int y = 0; y < Ny; y++)
		{
			for (int x = 0; x < Nx; x++)
			{
				int k = z * Nx * Ny + y * Nx + x;
				outputfile << setprecision(4) << x + 1 << "\t" << y + 1 << "\t " << z + 1 << "\t " << h_geo[k] << "\t " << output_rho[k] << "\t " << output_ux[k] << "\t " << output_uy[k] << "\t " << output_uz[k] << endl;
			}
		}
	}
	outputfile.close();
}

void MRTGPU::freemem()
{
	free(h_geo);
	free(h_f); free(h_f_post);
	free(h_rho);
	free(h_ux); free(h_uy); free(h_uz);
	free(output_rho);
	free(output_ux); free(output_uy); free(output_uz);

	hipFree(d_geo);
	hipFree(d_f); hipFree(d_f_post); hipFree(d_feq);
	hipFree(m_f); hipFree(m_eq); hipFree(inv_f); hipFree(inv_feq);
	hipFree(d_rho);
	hipFree(d_ux);	hipFree(d_uy);	hipFree(d_uz);
	hipFree(s_xx); hipFree(s_xy); hipFree(s_xz);
	hipFree(s_yx); hipFree(s_yy); hipFree(s_yz);
	hipFree(s_zx); hipFree(s_zy); hipFree(s_zz);
	hipFree(sigma);
}